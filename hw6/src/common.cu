#include "common.h"
#include <stdio.h>


hipError_t checkCuda(hipError_t result) {
        if (result != hipSuccess) {
            fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
            exit(-1);
        }
        return result;
}
